#include "hip/hip_runtime.h"
//
//  pole.cu
//  pole
//
//  Created by Dwight Bell on 8/18/10.
//  Copyright dbelll 2010. All rights reserved.
//

#include <hip/hip_runtime.h>
#include "cutil.h"

#include "cuda_rand.cu"

#include "pole.h"
#include "cuda_utils.h"

#define BLOCK_SIZE 256

static PARAMS _p;

static int __iTemp;
static float __fTemp;

// private prototypes
void display_help();

COMMON_VALS *initialize_common_values()
{
	COMMON_VALS *cv = (COMMON_VALS *)malloc(sizeof(COMMON_VALS));
	cv->seeds = (unsigned *)malloc(_p.agents * 4 * sizeof(unsigned));
	cv->agent_states = (float *)malloc(_p.agents * _p.state_size * sizeof(float));
	cv->last_action = (float *)malloc(_p.agents * sizeof(float));
	return cv;
}

void free_common_values(COMMON_VALS *cv)
{
	if (cv) {
		if (cv->seeds) free(cv->seeds);
		if (cv->agent_states) free(cv->agent_states);
		if (cv->last_action) free(cv->last_action);
		free(cv);
	}
}

void transfer_to_device(COMMON_VALS *cv)
{
	// tra
}

RESULTS *allocate_result_arrays()
{
	RESULTS *r = (RESULTS *)malloc(sizeof(RESULTS));
	r->results = (float *)malloc(_p.agents * sizeof(float));
	return r;
}

void free_result_arrays(RESULTS *r)
{
	if (r) {
		if (r->results) free(r->results);
		free(r);
	}
}

void run_GPU(COMMON_VALS *cv, RESULTS *r)
{
	
}

void run_CPU(COMMON_VALS *cv, RESULTS *r)
{
}

void read_params(int argc, const char **argv)
{
	if (argc == 1 || PARAM_PRESENT("HELP")) { display_help(); exit(1); }
	
	_p.trials = GET_PARAM("TRIALS", 1024);
	_p.agent_group_size = GET_PARAM("AGENT_GROUP_SIZE", 32);
	_p.block_sharing = (_p.agent_group_size >= 2);
	_p.agents = _p.trials * _p.agent_group_size;
	_p.time_steps = GET_PARAM("TIME_STEPS", 64);
	_p.sharing_interval = GET_PARAM("SHARING_INTERVAL", 4);
	if (0 != _p.time_steps % _p.sharing_interval){
		printf("Inconsistent arguments: TIME_STEPS=%d, SHARING_INTERVAL=%d\n", 
				_p.time_steps, _p.sharing_interval);
		exit(1);
	}
	_p.num_sharing_intervals = _p.time_steps / _p.sharing_interval;
	_p.data_lines = GET_PARAM("DATA_LINES", 16);
	if (0 != _p.time_steps % _p.data_lines){
		printf("Inconsistent arguments: TIME_STEPS=%d, DATA_LINES=%d\n", 
				_p.time_steps, _p.data_lines);
		exit(1);
	}
	_p.data_interval = _p.time_steps / _p.data_lines;
	_p.epsilon = GET_PARAMF("EPSILON", .10f);
	
	if (0 != _p.time_steps % BLOCK_SIZE){
		printf("Inconsistent argument: TIME_STEPS=%d, not a multiple of BLOCKSIZE which is %d\n", 
				_p.time_steps, BLOCK_SIZE);
		exit(1);
	}
	_p.blocks = _p.time_steps / BLOCK_SIZE;
	_p.run_on_CPU = GET_PARAM("RUN_ON_CPU", 1);
	_p.run_on_GPU = GET_PARAM("RUN_ON_GPU", 1);
	_p.no_print = PARAM_PRESENT("NO_PRINT");
	
	_p.state_size = GET_PARAM("STATE_SIZE", 4);

	printf("[POLE][TRIALS%7d][TIME_STEPS%7d][SHARING_INTERVAL%7d][AGENT_GROUP_SIZE%7d]""[EPSILON%7.4f][DATA_LINES%7d][STATE_SIZE%7d]\n", 
			_p.trials, _p.time_steps, _p.sharing_interval, _p.agent_group_size, _p.epsilon, 
			_p.data_lines, _p.state_size);
}

void display_help()
{
	printf("bandit parameters:\n");
	printf("  --TRIALS              number of trials for averaging reults\n");
	printf("  --AGENT_GROUP_SIZE    size of agent groups that will communicate\n");
	printf("  --TIME_STEPS          total number of time steps for each trial\n");
	printf("  --SHARING_INTERVAL    number of time steps between agent communication\n");
	printf("  --DATA_LINES          number of data samples in the report\n");
	printf("  --EPSILON             float value for epsilon\n");
	printf("  --RUN_ON_GPU          1 = run on GPU, 0 = do not run on GPU\n");
	printf("  --RUN_ON_CPU          1 = run on CPU, 0 = do not run on CPU\n");
	printf("  --HELP                print this help message\n");
	printf("default values will be used for any parameters not on command line\n");
}

#pragma mark CPU
__host__ void cpu_operation(int n, float *x)
{
	unsigned int timer;
	CREATE_TIMER(&timer);
	START_TIMER(timer);
	for (int i = 0; i < n; i++) {
		x[i] = sqrt(x[i]);
	}
	STOP_TIMER(timer, "cpu operation");
}

#pragma mark GPU
__global__ void kernel_operation(int n, float *x)
{
	int i = threadIdx.x + (blockIdx.x * blockDim.x) + (blockIdx.y * gridDim.x * blockDim.x);
	if (i >= n) return;
	
	x[i] = sqrt(x[i]);
}

void gpu_operation(int n, float *x)
{
	unsigned int timer;
	CREATE_TIMER(&timer);

	// copy data to device
	START_TIMER(timer);
	float *d_x = NULL;
	int size = n * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, size));
	CUDA_SAFE_CALL(hipMemcpy(d_x, x, size, hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	STOP_TIMER(timer, "copy data to device");


	// calculate size of block and grid
	int tot_threads = 32 * (1 + (n-1)/32);	// smallest multiple of 32 >= n
	dim3 blockDim(min(512,tot_threads));
	dim3 gridDim(1 + (tot_threads -1)/512, 1);
	if (gridDim.x > 65535) {
		gridDim.y = 1 + (gridDim.x-1) / 65535;
		gridDim.x = 1 + (gridDim.x-1) / gridDim.y;
	}
	
	// Do the operation on the device
	START_TIMER(timer);	
	kernel_operation<<<gridDim, blockDim>>>(n, d_x);
	hipDeviceSynchronize();
	STOP_TIMER(timer, "GPU operations");

	// Check if kernel execution generated an error
	CUT_CHECK_ERROR("Kernel execution failed");
	
	
	// copy results back to host
	START_TIMER(timer);
	CUDA_SAFE_CALL(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
	STOP_TIMER(timer, "copy data back to host");
	
	
}