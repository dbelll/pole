#include "hip/hip_runtime.h"
//
//  pole.cu
//  pole
//
//  Created by Dwight Bell on 8/18/10.
//  Copyright dbelll 2010. All rights reserved.
//

#include <hip/hip_runtime.h>
#include "cutil.h"

#include "cuda_rand.cu"

#include "pole.h"
#include "cuda_utils.h"
#include "main.h"
#include "cuda_row_reduction.h"

// paramaters in constant memory on the device
__constant__ unsigned dc_agents;
__constant__ unsigned dc_time_steps;

__constant__ float dc_epsilon;
__constant__ float dc_gamma;
__constant__ float dc_lambda;
__constant__ float dc_alpha;

__constant__ unsigned dc_num_actions;
__constant__ unsigned dc_num_features;

__constant__ unsigned dc_test_interval;
__constant__ unsigned dc_test_reps;

__constant__ unsigned dc_start_time;
__constant__ unsigned dc_end_time;


// fixed pointers in constant memory on the device
__constant__ unsigned *dc_seeds;
__constant__ float *dc_theta;
__constant__ float *dc_e;
__constant__ float *dc_s;
__constant__ float *dc_Q;
__constant__ unsigned *dc_action;

static AGENT_DATA *last_CPU_agent_dump;
static unsigned *d_seeds;
static float *d_theta;
static float *d_e;
static float *d_s;
static float *d_Q;
static unsigned *d_action;

void set_start_end_times(unsigned start, unsigned end)
{
//	printf("pole_kernel start=%d, end=%d\n", start, end);
//	fflush(NULL);

	hipMemcpyToSymbol(HIP_SYMBOL("dc_start_time"), &start, sizeof(unsigned));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_end_time"), &end, sizeof(unsigned));
}

// copy parameter values to constant memory on device
void set_constant_params(PARAMS p)
{
	hipMemcpyToSymbol(HIP_SYMBOL("dc_agents"), &p.agents, sizeof(unsigned));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_time_steps"), &p.time_steps, sizeof(unsigned));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_epsilon"), &p.epsilon, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_gamma"), &p.gamma, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_lambda"), &p.lambda, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_alpha"), &p.alpha, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_num_actions"), &p.num_actions, sizeof(unsigned));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_num_features"), &p.num_features, sizeof(unsigned));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_test_interval"), &p.test_interval, sizeof(unsigned));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_test_reps"), &p.test_reps, sizeof(unsigned));
}

// copy agent data pointers (device pointers) to constant memory on device
void set_constant_pointers(AGENT_DATA *ag)
{
	hipMemcpyToSymbol(HIP_SYMBOL("dc_seeds"), &ag->seeds, sizeof(unsigned *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_theta"), &ag->theta, sizeof(float *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_e"), &ag->e, sizeof(float *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_s"), &ag->s, sizeof(float *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_Q"), &ag->Q, sizeof(float *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_action"), &ag->action, sizeof(unsigned *));
}


/*
	Procedures for setting up and running the pole balancing experiements on CPU and GPU
 */

static PARAMS _p;
static unsigned g_seeds[4] = {2784565659u, 1491908209u, 3415062841u, 3293636241u};

#pragma mark CPU & GPU

// random number in an interval from -max to +max using random uniform distribution
__host__ __device__ float random_interval(unsigned *seeds, unsigned stride, float max)
{
#pragma unused(sd)
	float r = (-max) + 2 * max * RandUniform(seeds, stride);
	// keep generating values until one is within -max to +max
//	do {
//		r = RandNorm(seeds, stride) * sd;
//	} while (r < -max || r > max);
	return r;
}

// randomize the state
__host__ __device__ void randomize_state(float *s, unsigned *seeds, unsigned stride)
{
	s[0] = random_interval(seeds, stride, ANGLE_MAX);
//	s[stride] = random_interval(seeds, stride, ANGLE_VEL_MAX);
	s[stride] = 0.0f;
	s[2*stride] = random_interval(seeds, stride, X_MAX);
//	s[3*stride] = random_interval(seeds, stride, X_VEL_MAX);
	s[3*stride] = 0.0f;
}

// reset eligibility traces to 0.0f
__host__ __device__ void reset_trace(float *e, unsigned num_features, unsigned num_actions, 
										unsigned stride)
{
	for (int f = 0; f < num_features; f++) {
		for (int a = 0; a < num_actions; a++) {
			e[(a + f * num_actions) * stride] = 0.0f;
		}
	}
}

__device__ void reset_traceGPU(float *e)
{
	for (int f = 0; f < dc_num_features; f++) {
		for (int a = 0; a < dc_num_actions; a++) {
			e[(a + f * dc_num_actions) * dc_agents] = 0.0f;
		}
	}
}

__device__ __host__ unsigned terminal_state(float *s, unsigned stride)
{
	return s[2*stride] < X_MIN || s[2*stride] > X_MAX || 
			s[0] < ANGLE_MIN || s[0] > ANGLE_MAX;
}


// take an action from the current state, s, returning the reward and saving the new state in s_prime
__device__ __host__ float take_action(unsigned a, float *s, float *s_prime, unsigned stride)
{
	// formulas are from: Brownlee. The pole balancing problem: a benchmark control theory problem. hdl.handle.net (2005)
	
	// determine force from the action
	float F = a ? FORCE : -FORCE;

	float ang = s[0];
	float ang_vel = s[stride];
	float cos_a = cos(ang);
	float sin_a = sin(ang);
	
	// calculate angular acceleration
	float ang_accel = GRAV * sin_a;
	ang_accel += cos_a * (-F - POLE_MASS * POLE_LENGTH * ang_vel * ang_vel * sin_a) / 
							(CART_MASS + POLE_MASS);
	ang_accel /= POLE_LENGTH * (4.0f/3.0f - POLE_MASS * cos_a * cos_a / (CART_MASS + POLE_MASS));
	
	float x = s[2*stride];
	float x_vel = s[3*stride];

	// calculate x acceleration
	float x_accel = F + POLE_MASS * POLE_LENGTH * (ang_vel * ang_vel * sin_a - ang_accel * cos_a);
	x_accel /= (CART_MASS + POLE_MASS);
	
	// update ang, ang_vel and x, x_vel
	s_prime[0] = ang + TAU * ang_vel;
	s_prime[stride] = ang_vel + TAU * ang_accel;
	s_prime[2*stride] = x + TAU * x_vel;
	s_prime[3*stride] = x_vel + TAU * x_accel;
	
	// determine the reward
	float reward = terminal_state(s_prime, stride) ? REWARD_FAIL : REWARD_NON_FAIL;
	
	return reward;
}

//float take_action_debug(unsigned a, float *s, float *s_prime, unsigned stride)
//{
//	// formulas are from: Brownlee. The pole balancing problem: a benchmark control theory problem. hdl.handle.net (2005)
//	// determine force from the action
//	float F = a ? FORCE : -FORCE;
//
//	float ang = s[0];
//	float ang_vel = s[stride];
//	float x = s[2*stride];
//	float x_vel = s[3*stride];
//
//	float cos_a = cos_a(ang);
//	float sin_a = sin_a(ang);
//	
//	printf("[tack_action_debug] action=%d, angle=%7.4f, angle_vel=%7.4f\n", a, 
//
//	// calculate angular acceleration
//	float ang_accel = GRAV * sin_a;
//	ang_accel += cos_a * (-F - POLE_MASS * POLE_LENGTH * ang_vel * ang_vel * sin_a) / 
//							(CART_MASS + POLE_MASS);
//	ang_accel /= POLE_LENGTH * (4.0f/3.0f - POLE_MASS * cos_a * cos_a / (CART_MASS + POLE_MASS));
//	
//	// calculate x acceleration
//	float x_accel = F + POLE_MASS * POLE_LENGTH * (ang_vel * ang_vel * sin_a - ang_accel * cos_a);
//	x_accel /= (CART_MASS + POLE_MASS);
//	
//	// update ang, ang_vel and x, x_vel
//	s_prime[0] = ang + TAU * ang_vel;
//	s_prime[1] = ang_vel + TAU * ang_accel;
//	s_prime[2] = x + TAU * x_vel;
//	s_prime[3] = x_vel + TAU * x_accel;
//	
//	// determine the reward
//	float reward = REWARD_NON_FAIL;
//	if (s_prime[2] < X_MIN || s_prime[2] > X_MAX || 
//		s_prime[0] < ANGLE_MIN || s_prime[0] > ANGLE_MAX) 
//	{
//		reward = REWARD_FAIL;
//	}
//	
//	return reward;
//}

// Calculate which feature division the state value falls into, based on the min, max,
// and number of divisions.
__device__ __host__ unsigned feature_val_for_state_val(float s, float minv, float maxv, 
														unsigned div)
{
	return max(0, min(div-1, (unsigned)((s-minv)/(maxv-minv) * (float)div)));
}

// Determine which feature corresponds to the given state
__device__ __host__ unsigned feature_for_state(float *s, unsigned stride)
{
	unsigned feature = feature_val_for_state_val(s[0], ANGLE_MIN, ANGLE_MAX, ANGLE_DIV);
	feature += (ANGLE_DIV) * 
				feature_val_for_state_val(s[stride], ANGLE_VEL_MIN, ANGLE_VEL_MAX, ANGLE_VEL_DIV);
	feature += (ANGLE_DIV * ANGLE_VEL_DIV) * 
				feature_val_for_state_val(s[2 * stride], X_MIN, X_MAX, X_DIV);
	feature += (ANGLE_DIV * ANGLE_VEL_DIV * X_DIV) * 
				feature_val_for_state_val(s[3 * stride], X_VEL_MIN, X_VEL_MAX, X_VEL_DIV);
	return feature;
}

__device__ __host__ const char * failure_type(float *s, unsigned stride)
{
	if (s[0] < ANGLE_MIN) return "Angle < MIN";
	if (s[0] > ANGLE_MAX) return "Angle > MAX";
	if (s[2] < X_MIN) return "X < MIN";
	if (s[2] > X_MAX) return "X > MAX";
	return "";
}

// Calculate a number with the division for each state variable
__device__ __host__ unsigned divs_for_feature(unsigned feature)
{
	unsigned divs = feature % ANGLE_DIV;
	feature /= ANGLE_DIV;
	divs += 16 * (feature % ANGLE_VEL_DIV);
	feature /= ANGLE_VEL_DIV;
	divs += 256 * (feature % X_DIV);
	feature /= X_DIV;
	divs += 4096 * feature;
	return divs;
}

// calculate the Q value for an action from a state
__device__ __host__ float calc_Q(float *s, unsigned a, float *theta, unsigned stride, 
																			unsigned num_actions)
{
	// only one feature corresponds with any given state
	unsigned feature = feature_for_state(s, stride);
	float Q = theta[(a + feature * num_actions) * stride];
	return Q;
}

/*
	Calculate the Q value for a given state/action on GPU
	Stride is assumed to be BLOCK_SIZE for s and a and dc_agents for theta
*/
__device__ float calc_QGPU(float *s, unsigned a, float *theta)
{
	// only one feature corresponds with any given state
	unsigned feature = feature_for_state(s, BLOCK_SIZE);
	float Q = theta[(a + feature * dc_num_actions) * dc_agents];
	return Q;
}

__device__ __host__ void update_stored_Q(float *Q, float *s, float *theta, unsigned stride, 
																			unsigned num_actions)
{
	for (int a = 0; a < num_actions; a++) {
		Q[a * stride] = calc_Q(s, a, theta, stride, num_actions);
	}
}

__device__ void update_stored_QGPU(float *Q, float *s, float *theta)
{
	for (int a = 0; a < dc_num_actions; a++) {
		Q[a * BLOCK_SIZE] = calc_QGPU(s, a, theta);
	}
}

// Calculate the Q value for each action from the given state, storing the values in Q
// Return the action with the highest Q value
__device__ __host__ unsigned best_action(float *s, float *theta, float *Q, unsigned stride,
										 unsigned num_actions)
{
	// calculate the Q value for each action
	Q[0] = calc_Q(s, 0, theta, stride, num_actions);
	unsigned best_action = 0;
	float bestQ = Q[0];

	for (int a = 1; a < num_actions; a++) {
		Q[a * stride] = calc_Q(s, a, theta, stride, num_actions);
		if (Q[a * stride] > bestQ) {
			bestQ = Q[a * stride];
			best_action = a;
		}
	}
	return best_action;
}

/*
	Choose the best action on the GPU
	Strides are assumed to be BLOCK_SIZE for s and Q and dc_agents for theta
*/
__device__ unsigned best_actionGPU(float *s, float *theta, float *Q)
{
	// calculate the Q value for each action
	Q[0] = calc_QGPU(s, 0, theta);
	unsigned best_action = 0;
	float bestQ = Q[0];

	for (int a = 1; a < dc_num_actions; a++) {
		Q[a * BLOCK_SIZE] = calc_QGPU(s, a, theta);
		if (Q[a * BLOCK_SIZE] > bestQ) {
			bestQ = Q[a * BLOCK_SIZE];
			best_action = a;
		}
	}
	return best_action;
}

// choose action from current state, storing Q values for each possible action in Q
__device__ __host__ unsigned choose_action(float *s, float *theta, float epsilon, unsigned stride, 
											float *Q, unsigned num_actions, unsigned *seeds)
{
	// always calcualte the best action and store all the Q values for each action
	unsigned a = best_action(s, theta, Q, stride, num_actions);
	if (epsilon > 0.0f && RandUniform(seeds, stride) < epsilon){
		// choose random action
		float r = RandUniform(seeds, stride);
		a = r * num_actions;
	}
	return a;
}

/*
	choose action on GPU
	strides are assumed to be BLOCK_SIZE for s, Q, and seeds and dc_agents theta
*/
__device__ unsigned choose_actionGPU(float *s, float *theta, float *Q, unsigned *seeds)
{
	// always calcualte the best action and store all the Q values for each action
	unsigned a = best_actionGPU(s, theta, Q);
	if (dc_epsilon > 0.0f && RandUniform(seeds, BLOCK_SIZE) < dc_epsilon){
		// choose random action
		float r = RandUniform(seeds, BLOCK_SIZE);
		a = r * dc_num_actions;
	}
	return a;
}

// Update eligibility traces based on action and state
__host__ __device__ void update_trace(unsigned action, float *s, float *e, unsigned num_features,
										unsigned num_actions, unsigned stride, float gamma, float lambda)
{
	unsigned feature = feature_for_state(s, stride);
	for (int f = 0; f < num_features; f++) {
		for (int a = 0; a < num_actions; a++) {
			unsigned index = (a + f * num_actions) * stride;
			// Replacing trace with optional block
			if (f == feature) {
				// set to 1.0 for action selected from current state,
				// set to 0.0 for actions not taken from current state
				e[index] = (a == action) ? 1.0f : 0.0f;
			}else {
				// decay all other values
				e[index] *= gamma * lambda;
			}
		}
	}
}

/*
	Update the eligibility trace values on the GPU
	Stride is assumed to be BLOCK_SIZE for s and dc_agents for e
*/
__device__ void update_traceGPU(unsigned action, float *s, float *e)
{
	unsigned feature = feature_for_state(s, BLOCK_SIZE);
	for (int f = 0; f < dc_num_features; f++) {
		for (int a = 0; a < dc_num_actions; a++) {
			unsigned index = (a + f * dc_num_actions) * dc_agents;
			// Replacing trace with optional block
			if (f == feature) {
				// set to 1.0 for action selected from current state,
				// set to 0.0 for actions not taken from current state
				e[index] = (a == action) ? 1.0f : 0.0f;
			}else {
				// decay all other values
				e[index] *= dc_gamma * dc_lambda;
			}
		}
	}
}



// Update theta values for one agent
// theta = theta + alpha * delta * eligibility trace
__device__ __host__ void update_thetas(float *theta, float *e, float alpha, float delta, unsigned num_features, unsigned stride, unsigned num_actions)
{
	if (alpha == 0.0f || delta == 0.0f) return;
//#ifdef DUMP_THETA_UPDATE_CALCULATIONS
//	printf("updating thetas for alpha = %9.6f, delta = %9.6f\n", alpha, delta);
//#endif
	for (int fa = 0; fa < num_features * num_actions; fa++) {
		if (e[fa*stride] > 0.001f) {
//#ifdef DUMP_THETA_UPDATE_CALCULATIONS
//			printf("   feature-action %5d(%4x) %3d with trace %9.6f changed from %9.6f", (fa/num_actions), divs_for_feature(fa/num_actions), (fa%num_actions), e[fa*stride], theta[fa*stride]);
//#endif
			theta[fa * stride] += alpha * delta * e[fa * stride];
//#ifdef DUMP_THETA_UPDATE_CALCULATIONS
//			printf(" to %9.6f\n", theta[fa*stride]);
//#endif
		}
	}
}

/*
	Update thetas on GPU
	Stride is assumed to be dc_agents for theta and e
*/
__device__ void update_thetasGPU(float *theta, float *e, float delta)
{
	if (dc_alpha == 0.0f || delta == 0.0f) return;
	for (int fa = 0; fa < dc_num_features * dc_num_actions; fa++) {
	//**TODO** see if this test for size of e makes any difference, it was put here for printing purposes
		if (e[fa*dc_agents] > 0.001f) {
			theta[fa * dc_agents] += dc_alpha * delta * e[fa * dc_agents];
		}
	}
}

#pragma mark -
#pragma mark CPU

void set_params(PARAMS p){ _p = p;}

void dump_agent(AGENT_DATA *ag, unsigned agent)
{
	printf("[agent %d]: ", agent);
	printf("   seeds = %u, %u, %u, %u\n", ag->seeds[agent], ag->seeds[agent + _p.agents], 
									   ag->seeds[agent + 2*_p.agents], ag->seeds[agent + 3*_p.agents]);
#ifdef AGENT_DUMP_INCLUDE_THETA_E
	printf("FEATURE       ACTION    THETA       E  \n");
	for (int f = 0; f < _p.num_features; f++) {
		for (int action = 0; action < _p.num_actions; action++) {
			printf("%7d %4x %7d %9.6f %9.6f\n", f, divs_for_feature(f), action, 
				   ag->theta[agent + (action + f * _p.num_actions) * _p.agents], 
				   ag->e[agent + (action + f * _p.num_actions) * _p.agents]);
		}
	}
#endif
	printf("   angle    angleV       x         xV        Q0        Q1   feature\n");
	unsigned feature = feature_for_state(ag->s + agent, _p.agents);
	printf("%9.6f %9.6f %9.6f %9.6f %9.6f %9.6f %7d(%4x)\n", ag->s[agent], ag->s[agent + _p.agents], ag->s[agent + 2*_p.agents], ag->s[agent + 3*_p.agents], ag->Q[agent], ag->Q[agent + _p.agents],
		feature, divs_for_feature(feature));

	printf("chosen action is %d\n", ag->action[agent]);
	
	printf("ACTION  Q-value\n");
//		printf("number of actions is %d\n", p.num_actions);
	for (int action = 0; action < _p.num_actions; action++) {
		(action == ag->action[agent]) ? printf("-->") : printf("   ");
		printf("%3d  %9.6f\n", action, ag->Q[agent + action * _p.agents]);
	}
	printf("\n");
}

void dump_agents(const char *str, AGENT_DATA *ag)
{
	last_CPU_agent_dump = ag;
	printf("%s\n", str);
	for (int agent = 0; agent < _p.agents; agent++) {
		dump_agent(ag, agent);
	}
}

// generate random seeds for the sepecified number of agents
unsigned *create_seeds(unsigned num_agents)
{
	unsigned *seeds = (unsigned *)malloc(num_agents * 4 * sizeof(unsigned));
	for (int i = 0; i < num_agents * 4; i++) {
		seeds[i] = RandUniformui(g_seeds, 1);
	}
	return seeds;
}

// create wgts set initially to random values between RAND_WGT_MIN and RAND_WGT_MAX
float *create_theta(unsigned num_agents, unsigned num_features, unsigned num_actions)
{
#ifdef VERBOSE
	printf("create_theta for %d agents and %d features\n", num_agents, num_features);
#endif
	float *theta = (float *)malloc(num_agents * num_features * num_actions * sizeof(float));
	for (int i = 0; i < num_agents * num_features * num_actions; i++) {
//		float r = RandUniform(g_seeds, 1);
//		theta[i] = (RAND_WGT_MAX - RAND_WGT_MIN) * r + RAND_WGT_MIN;
//		printf("randome = %7.4f, theta = %7.4f\n", r, theta[i]);
		theta[i] = (RAND_WGT_MAX - RAND_WGT_MIN) * RandUniform(g_seeds, 1) + RAND_WGT_MIN;
	}
	return theta;
}

// initial eligibility traces to 0.0f
float *create_e(unsigned num_agents, unsigned num_features, unsigned num_actions)
{
#ifdef VERBOSE
	printf("create_e for %d agents and %d features and %d actions\n", num_agents, num_features, num_actions);
#endif
	float *e = (float *)malloc(num_agents * num_features * num_actions * sizeof(float));
	for (int i = 0; i < num_agents * num_features * num_actions; i++) {
		e[i] = 0.0f;
	}
	return e;
}

// initial random states
float *create_states(unsigned num_agents, unsigned *seeds)
{
	float *states = (float *)malloc(num_agents * _p.state_size * sizeof(float));
	for (int i = 0; i < num_agents; i++) {
//		states[i] = random_interval(seeds + i, num_agents, ANGLE_MAX, STATE_SD);
//		states[i + num_agents] = random_interval(seeds+i, num_agents, ANGLE_VEL_MAX, STATE_SD);
//		states[i + 2 * num_agents] = random_interval(seeds+i, num_agents, X_MAX, STATE_SD);
//		states[i + 3 * num_agents] = random_interval(seeds+i, num_agents, X_VEL_MAX, STATE_SD);
		randomize_state(states + i, seeds + i, num_agents);
	}
	return states;
}

RESULTS *initialize_results()
{
#ifdef VERBOSE
	printf("initializing result arrays...\n");
#endif
	RESULTS *r = (RESULTS *)malloc(sizeof(RESULTS));
	r->avg_fail = (float *)malloc((_p.time_steps / _p.test_interval) * sizeof(float));
	return r;
}

void free_results(RESULTS *r)
{
#ifdef VERBOSE
	printf("freeing result arrays...\n");
#endif
	if (r) {
		if (r->avg_fail) free(r->avg_fail);
		free(r);
	}
}

void display_results(const char *str, RESULTS *r)
{
	printf("%s \n", str);
	printf("    TEST  Avg Fails\n");
	for (int i = 0; i < _p.num_tests; i++) {
		printf("   [%4d]%9.4f\n", i, r->avg_fail[i]);
	}
}

unsigned *create_actions(unsigned num_agents, unsigned num_actions)
{
	unsigned *actions = (unsigned *)malloc(num_agents * num_actions * sizeof(unsigned));
	for (int i = 0; i < num_agents * num_actions; i++) {
		actions[i] = num_actions;	// not possible action
	}
	return actions;
}

// Initialize agents on the CPU.  Some values will be re-used for GPU agents
AGENT_DATA *initialize_agentsCPU()
{
#ifdef VERBOSE
	printf("initializing agents on CPU...\n");
#endif
	AGENT_DATA *ag = (AGENT_DATA *)malloc(sizeof(AGENT_DATA));
	ag->seeds = create_seeds(_p.agents);
	ag->theta = create_theta(_p.agents, _p.num_features, _p.num_actions);
	ag->e = create_e(_p.agents, _p.num_features, _p.num_actions);
//	unsigned rows = _p.agents * ((_p.state_size + 2) * _p.sharing_interval + _p.state_size + 1);
//	ag->ep_data = (float *)malloc(rows * sizeof(float));
	ag->s = create_states(_p.agents, ag->seeds);
	ag->Q = (float *)malloc(_p.agents * _p.num_actions * sizeof(float));
	ag->action = create_actions(_p.agents, _p.num_actions);
	return ag;
}

void dump_state(float *s, unsigned stride)
{
	printf("(%9.6f,%9.6f,%9.6f,%9.6f)[%d]\n", s[0], s[stride], s[2*stride], s[3*stride], 
															feature_for_state(s, stride));
}

// run tests for all agents and return the average failures
float run_test(AGENT_DATA *ag)
{
	unsigned num_failures = 0;
	
	// initialize all agent states
	for (int agent = 0; agent < _p.agents; agent++) {
//		printf("agent %d before testing...\n", agent);
//		dump_agent(ag, agent);
//		unsigned old_num_failures = num_failures;
		
		// save agent state prior to testing
		float s0 = ag->s[agent];
		float s1 = ag->s[agent + _p.agents];
		float s2 = ag->s[agent + 2*_p.agents];
		float s3 = ag->s[agent + 3*_p.agents];
		unsigned act = ag->action[agent];
		unsigned seed0 = ag->seeds[agent];
		unsigned seed1 = ag->seeds[agent + _p.agents];
		unsigned seed2 = ag->seeds[agent + 2 * _p.agents];
		unsigned seed3 = ag->seeds[agent + 3 * _p.agents];
		float Q0 = ag->Q[agent];
		float Q1 = ag->Q[agent + _p.agents];
		
//		randomize_state(ag->s + agent, ag->seeds + agent, _p.agents);
		ag->action[agent] = best_action(ag->s + agent, ag->theta + agent, ag->Q + agent, _p.agents, _p.num_actions);
//		choose_action(ag->s + agent, ag->theta + agent, 0.0f, _p.agents, ag->Q + agent, 
//																_p.num_actions, ag->seeds + agent);

		// run the test for specified number of reps
		for (int t = 0; t < _p.test_reps; t++) {
			take_action(ag->action[agent], ag->s+agent, ag->s+agent, _p.agents);
			if (terminal_state(ag->s + agent, _p.agents)){
				++num_failures;
				randomize_state(ag->s + agent, ag->seeds + agent, _p.agents);
			}
			// choose best action
			ag->action[agent] = best_action(ag->s + agent, ag->theta + agent, ag->Q + agent, _p.agents, _p.num_actions);
		}
		
		// restore agent state
		ag->s[agent] = s0;
		ag->s[agent + _p.agents] = s1;
		ag->s[agent + 2*_p.agents] = s2;
		ag->s[agent + 3*_p.agents] = s3;
		act = ag->action[agent] = act;
		ag->seeds[agent] = seed0;
		ag->seeds[agent + _p.agents] = seed1;
		ag->seeds[agent + 2 * _p.agents] = seed2;
		ag->seeds[agent + 3 * _p.agents] = seed3;
		ag->Q[agent] = Q0;
		ag->Q[agent + _p.agents] = Q1;
		
//		printf("after testing...\n");
//		dump_agent(ag, agent);

//		printf("agent %d failues = %d\n", agent, num_failures - old_num_failures);
	}



	return num_failures / (float)_p.agents;
}

void run_CPU_noshare(AGENT_DATA *ag, RESULTS *r)
{
	unsigned tot_fails = 0;
#ifdef DUMP_INTERMEDIATE_FAIL_COUNTS
	unsigned prev_tot_fails = 0;
#endif
#ifdef VERBOSE
	printf(" no sharing\n");
#endif

	// on entry the agent's theta, eligibility trace, and state values have been initialized
	
#ifdef DUMP_AGENT_ACTIONS
		printf("-----------------------------------------------------------\n");
		printf("---------------------- INITIAL SETUP ----------------------\n");
		printf("-----------------------------------------------------------\n");
#endif

	// set-up agents to begin the loop by choosing the first action and updating traces
	for (int agent = 0; agent < _p.agents; agent++) {
		ag->action[agent] = choose_action(ag->s + agent, ag->theta + agent, _p.epsilon, _p.agents,
										ag->Q + agent, _p.num_actions, ag->seeds + agent);

#ifdef DUMP_AGENT_ACTIONS
		printf("agent %d will choose action %d from state ", agent, ag->action[agent]);
		dump_state(ag->s + agent, _p.agents);
#endif

		update_trace(ag->action[agent], ag->s + agent, ag->e + agent, _p.num_features, 
												_p.num_actions, _p.agents, _p.gamma, _p.lambda);		
	}

#ifdef DUMP_AGENT_ACTIONS
	printf("----------------------------------------------------\n");
	printf("-------------- BEGIN MAIN LOOP ---------------------\n");
	printf("----------------------------------------------------\n");
#endif	

	int k = 1;
	if (_p.time_steps > 40) {
		k = 1 + (_p.time_steps-1)/40;
	}
	
	for (int i = 0; i < (_p.time_steps / k); i++) {
		printf("-");
	}
	printf("|\n");


	// main loop, repeat for the number of trials
	for (int t = 0; t <= _p.time_steps; t++) {
		if (0 == (t+1) % k) {
			printf(".");
			fflush(NULL);
		}

		if (0 == (t % _p.test_interval) && (t > 0)) {
			// run the test and store the result
			unsigned iTest = (t-1) / _p.test_interval;
			r->avg_fail[iTest] = run_test(ag);
//			printf("*********[%3d] test results =%7.2f\n", iTest, r->avg_fail[iTest]);
		}
		if (t == _p.time_steps) break;

#ifdef DUMP_AGENT_ACTIONS
	printf("\n------------------ TIME STEP%3d ------------------------\n", t);
#endif	

		for (int agent = 0; agent < _p.agents; agent++) {

			// stored state is s      stored Q's are Q(s)  
			
#ifdef DUMP_AGENT_ACTIONS
			printf("<<<<<<<< AGENT %d >>>>>>>>>>>>\n", agent);
			printf("time step %d, agent %d ready for next action\n", t, agent);
			dump_agent(ag, agent);
#endif
			// take the action already chosen and saved in ag->action
			unsigned prev_feature = feature_for_state(ag->s, _p.agents);
			float reward = take_action(ag->action[agent], ag->s + agent, ag->s + agent, _p.agents);

#ifdef DUMP_AGENT_BRIEF
			(agent == 0) ? printf("[step%4d]", t) : printf("          ");
			printf("[agent%3d] took action:%2d, got reward:%6.3f, new state is ", agent, ag->action[agent], reward);
			dump_state(ag->s + agent, _p.agents);
#endif
			
			// stored state is s_prime      stored Q's are Q(s)
			unsigned fail = terminal_state(ag->s + agent, _p.agents);
			if (fail){
#ifdef DUMP_FAILURE_TIMES
				printf("Agent%4d Failure at %d taking action %d from state %d (%x) resulting in %s\n", agent, t, ag->action[agent], prev_feature, divs_for_feature(prev_feature), failure_type(ag->s + agent, _p.agents));
#endif
#ifdef DUMP_AGENT_STATE_ON_FAILURE
				printf("session initial state was angle=%6.2f,  angleV=%6.2f, x=%6.2f, xV=%6.2f\n",
						orig_a, orig_aV, orig_x, orig_xV);
				dump_agent(ag, agent);
#endif
				randomize_state(ag->s + agent, ag->seeds + agent, _p.agents);
				
//				if (agent == 0){
//					orig_a = ag->s[0];
//					orig_aV = ag->s[_p.agents];
//					orig_x = ag->s[2*_p.agents];
//					orig_xV = ag->s[3*_p.agents];
//				}
				++tot_fails;
			}
						

			float Q_a = ag->Q[agent + ag->action[agent] * _p.agents];

#ifdef DUMP_AGENT_ACTIONS
			if (fail) printf("-------------------------------------------------------\n!!!! terminal state reached, next state is random\n---------------------------------------------------\n\n");
			printf("agent %d, took action %d, got reward %6.3f, now in state s_prime = " , agent,	ag->action[agent], reward);
			dump_state(ag->s + agent, _p.agents);
#endif

#ifdef DUMP_CALCULATIONS
			printf("reward is %9.6f, Q[%d] for state s is %9.6f\n", reward, ag->action[agent], Q_a);
#endif

//			ag->prev_action[agent] = ag->action[agent];
//			ag->f_prev_state[agent] = feature_for_state(ag->s + agent, _p.agents);
			ag->action[agent] = choose_action(ag->s + agent, ag->theta + agent, _p.epsilon,
								_p.agents, ag->Q + agent, _p.num_actions, ag->seeds + agent);
			
			// Stored Q values are now based on the new state, s_prime

#ifdef DUMP_AGENT_ACTIONS
			printf("agent %d's next action will be %d with Q-value %9.6f\n", agent, ag->action[agent], ag->Q[agent + ag->action[agent] * _p.agents]);
//			dump_state(ag->s + agent, _p.agents);
#endif

			float Q_a_prime = ag->Q[agent + ag->action[agent] * _p.agents];
			float delta = reward - Q_a + (fail ? 0 : _p.gamma * Q_a_prime);

#ifdef DUMP_CALCULATIONS
			printf("discount is %9.6f, newQ[%d] is %9.6f, so delta is %9.6f\n", _p.gamma, 
												ag->action[agent], (fail ? 0.0f : Q_a_prime), delta);
#endif

#ifdef DUMP_AGENT_ACTIONS
			printf("[update_theta]:\n");
#endif

			update_thetas(ag->theta + agent, ag->e + agent, _p.alpha, delta, _p.num_features,
																	 _p.agents, _p.num_actions);
			if (fail) reset_trace(ag->e + agent, _p.num_features, _p.num_actions, _p.agents);

			update_stored_Q(ag->Q + agent, ag->s + agent, ag->theta + agent, _p.agents, 
																				_p.num_actions);
			
#ifdef DUMP_AGENT_ACTIONS
			printf("[update_trace]\n");
#endif

			update_trace(ag->action[agent], ag->s + agent, ag->e + agent, _p.num_features, _p.num_actions, _p.agents, _p.gamma, _p.lambda);
			
#ifdef DUMP_AGENT_ACTIONS
//			printf("agent state after updating theta and eligibility trace:\n");
//			dump_agent(ag, agent);
#endif
		}

#ifdef DUMP_INTERMEDIATE_FAIL_COUNTS
		if (0 == (1+t) % _p.test_interval) {
			printf("intermediate fail count =%7.2f\n", (tot_fails - prev_tot_fails)/(float)_p.trials);
			prev_tot_fails = tot_fails;
		}
#endif


	}
	
//	printf("*********[%3d] test results =%7.2f\n", _p.time_steps / _p.test_interval, run_test(ag));

#ifdef DUMP_TERMINAL_AGENT_STATE
	printf("\n----------------------------------------------\n");
	dump_agents("               ENDING AGENT STATES\n", ag);
#endif		
	printf("total failures = %d\n", tot_fails);
}

void run_CPU_share(AGENT_DATA *cv, RESULTS *r)
{
#ifdef VERBOSE
	printf(" sharing in agent blocks of %d\n", _p.agent_group_size);
#endif

}

void run_CPU(AGENT_DATA *ag, RESULTS *r)
{
#ifdef VERBOSE
	printf("\n==============================================\nrunning on CPU...\n");
#endif

#ifdef DUMP_INITIAL_AGENTS
	dump_agents("Initial agents on CPU", ag);
#endif
	unsigned timer;
	CREATE_TIMER(&timer);
	START_TIMER(timer);
	_p.agent_group_size > 1 ? run_CPU_share(ag, r) : run_CPU_noshare(ag, r);	
	STOP_TIMER(timer, "run on CPU");
}

void free_agentsCPU(AGENT_DATA *ag)
{
#ifdef VERBOSE
	printf("freeing agents on CPU...\n");
#endif
	if (ag) {
		if (ag->seeds) free(ag->seeds);
		if (ag->theta) free(ag->theta);
		if (ag->e) free(ag->e);
		if (ag->s) free(ag->s);
		if (ag->Q) free(ag->Q);
		if (ag->action) free(ag->action);
		free(ag);
	}
}

#pragma mark -
#pragma mark GPU

AGENT_DATA *copy_GPU_agents()
{
	AGENT_DATA *agGPUcopy = (AGENT_DATA *)malloc(sizeof(AGENT_DATA));
	agGPUcopy->seeds = host_copyui(d_seeds, _p.agents * 4);
	agGPUcopy->theta = host_copyf(d_theta, _p.agents * _p.num_features * _p.num_actions);
	agGPUcopy->e = host_copyf(d_e, _p.agents * _p.num_features * _p.num_actions);
	agGPUcopy->s = host_copyf(d_s, _p.agents * _p.state_size);
	agGPUcopy->Q = host_copyf(d_Q, _p.agents * _p.num_actions);
	agGPUcopy->action = host_copyui(d_action, _p.agents);
	return agGPUcopy;
}

// check if s1[i] and s2[i] within small value of each other
unsigned mismatch(float *s1, float *s2, unsigned i)
{
	float small = 1.0e-4;
	return s1[i] > (s2[i]+small) || s1[i] < (s2[i]-small);
}

unsigned mismatchui(unsigned *s1, unsigned *s2, unsigned i)
{
	unsigned small = 0;
	return s1[i] > (s2[i]+small) || s1[i] < (s2[i]-small);
}

// check that the GPU agent information copied from the device is the same as the
// CPU agent information pointed to by last_CPU_agent_dump
void check_agents(AGENT_DATA *agGPUcopy)
{
	for (int agent = 0; agent < _p.agents; agent++) {
		printf("[agent%4d] ", agent);
		unsigned match = 1;

		for (int s = 0; s < 4; s++) {
			if (mismatchui(agGPUcopy->seeds, last_CPU_agent_dump->seeds, agent + s*_p.agents)){
				match = 0;
				printf("seed mismatch, ");
				break;
			}
			if (mismatch(agGPUcopy->s, last_CPU_agent_dump->s, agent + s*_p.agents)){
				match = 0;
				printf("state mismatch, ");
				break;
			}
		}
		
		
		
		for (int th = 0; th < _p.num_features * _p.num_actions; th++) {
			if (mismatch(agGPUcopy->theta, last_CPU_agent_dump->theta, agent + th*_p.agents)){
				match = 0;
				printf("theta mismatch feature=%d, action=%d, %f vs %f\n", th/_p.num_actions, th % _p.num_actions, agGPUcopy->theta[agent + th * _p.agents], last_CPU_agent_dump->theta[agent + th * _p.agents]);
//				break;
			}
			if (mismatch(agGPUcopy->e, last_CPU_agent_dump->e, agent + th*_p.agents)){
				match = 0;
				printf("trace mismatch feature=%d, action=%d\n", th/_p.num_actions, th % _p.num_actions);
//				break;
			}
		}
		
		printf(match ? "match\n" : "\n");
	}
}

void dump_agents_GPU(const char *str, unsigned check)
{
	AGENT_DATA *agGPUcopy = copy_GPU_agents();
	if (check) check_agents(agGPUcopy);
	dump_agents(str, agGPUcopy);
	free_agentsCPU(agGPUcopy);
}

/*
	Initializes agent data on GPU by copying the CPU data.
	Also initialized constant memory pointers to point to the GPU data.
	Allocates device memory for:
		dc_seeds, dc_theta, dc_e, dc_s, dc_Q, and dc_action
	Device pointers also stored in host memory: d_seeds, d_theta, d_e, d_s, d_Q, and d_action,
	which are used to free the device memory.
*/
void initialize_agentsGPU(AGENT_DATA *agCPU)
{
#ifdef VERBOSE
	printf("initializing agents on GPU...\n");
#endif
	d_seeds = device_copyui(agCPU->seeds, _p.agents * 4);
	d_theta = device_copyf(agCPU->theta, _p.agents * _p.num_features * _p.num_actions);
	d_e = device_copyf(agCPU->e, _p.agents * _p.num_features * _p.num_actions);
	d_s = device_copyf(agCPU->s, _p.agents * _p.state_size);
	d_Q = device_copyf(agCPU->Q, _p.agents * _p.num_actions);
	d_action = device_copyui(agCPU->action, _p.agents);
	
	hipMemcpyToSymbol(HIP_SYMBOL("dc_seeds"), &d_seeds, sizeof(unsigned *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_theta"), &d_theta, sizeof(float *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_e"), &d_e, sizeof(float *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_s"), &d_s, sizeof(float *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_Q"), &d_Q, sizeof(float *));
	hipMemcpyToSymbol(HIP_SYMBOL("dc_action"), &d_action, sizeof(unsigned *));
}

// free all agent data from GPU
void free_agentsGPU()
{
#ifdef VERBOSE
	printf("freeing agents on GPU...\n");
#endif
	if (d_seeds) hipFree(d_seeds);
	if (d_theta) hipFree(d_theta);
	if (d_e) hipFree(d_e);
	if (d_s) hipFree(d_s);
	if (d_Q) hipFree(d_Q);
	if (d_action) hipFree(d_action);
}

/*
	copy state information from global device memory to shared memory
	assumes stride is BLOCK_SIZE for shared memory and dc_agents for global memory
*/
#define COPY_STATE_TO_SHARED(iLocal, iGlobal)															\
			s_s[iLocal] = dc_s[iGlobal];										\
			s_s[iLocal + BLOCK_SIZE] = dc_s[iGlobal + dc_agents];				\
			s_s[iLocal + 2*BLOCK_SIZE] = dc_s[iGlobal + 2*dc_agents];			\
			s_s[iLocal + 3*BLOCK_SIZE] = dc_s[iGlobal + 3*dc_agents];			\
			s_action[iLocal] = dc_action[iGlobal];									\
			s_seeds[iLocal] = dc_seeds[iGlobal];								\
			s_seeds[iLocal + BLOCK_SIZE] = dc_seeds[iGlobal + dc_agents];		\
			s_seeds[iLocal + 2*BLOCK_SIZE] = dc_seeds[iGlobal + 2*dc_agents];	\
			s_seeds[iLocal + 3*BLOCK_SIZE] = dc_seeds[iGlobal + 3*dc_agents];	\
			s_Q[iLocal] = dc_Q[iGlobal];										\
			s_Q[iLocal + BLOCK_SIZE] = dc_Q[iGlobal + dc_agents];
			
#define COPY_STATE_TO_GLOBAL(iLocal, iGlobal)									\
			dc_s[iGlobal] = s_s[iLocal];										\
			dc_s[iGlobal + dc_agents] = s_s[iLocal + BLOCK_SIZE];				\
			dc_s[iGlobal + 2*dc_agents] = s_s[iLocal + 2*BLOCK_SIZE];			\
			dc_s[iGlobal + 3*dc_agents] = s_s[iLocal + 3*BLOCK_SIZE];			\
			dc_action[iGlobal] = s_action[iLocal];									\
			dc_seeds[iGlobal] = s_seeds[iLocal];								\
			dc_seeds[iGlobal + dc_agents] = s_seeds[iLocal + BLOCK_SIZE];		\
			dc_seeds[iGlobal + 2*dc_agents] = s_seeds[iLocal + 2*BLOCK_SIZE];	\
			dc_seeds[iGlobal + 3*dc_agents] = s_seeds[iLocal + 3*BLOCK_SIZE];	\
			dc_Q[iGlobal] = s_Q[iLocal];										\
			dc_Q[iGlobal + dc_agents] = s_Q[iLocal + BLOCK_SIZE];

__global__ void pole_kernel(float *results)
{
	unsigned iGlobal = threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
	unsigned idx = threadIdx.x;
	
	__shared__ float s_s[4 * BLOCK_SIZE];
	__shared__ unsigned s_action[BLOCK_SIZE];
	__shared__ unsigned s_seeds[4 * BLOCK_SIZE];
	__shared__ float s_Q[2*BLOCK_SIZE];
	
	COPY_STATE_TO_SHARED(idx, iGlobal);
	
	// prepare for first iteration by chosing first action and updating the trace
	if (dc_start_time == 0) {
//		dc_action[iGlobal] = choose_actionGPU(dc_s + iGlobal, dc_theta + iGlobal, dc_Q + iGlobal,
//																				 dc_seeds + iGlobal);
//		update_trace(dc_action[iGlobal], dc_s + iGlobal, dc_e + iGlobal, dc_num_features, 
//													dc_num_actions, dc_agents, dc_gamma, dc_lambda);
		s_action[idx] = choose_actionGPU(s_s + idx, dc_theta + iGlobal, s_Q + idx, s_seeds + idx);
		update_traceGPU(s_action[idx], s_s + idx, dc_e + iGlobal);
	}
	
	for (int t = dc_start_time; t <= dc_end_time; t++) {
		
		// run the test
		if (0 == (t % dc_test_interval) && (t > dc_start_time)){
			// run the test and record the results

			// save state to back to global memory
			COPY_STATE_TO_GLOBAL(idx, iGlobal);
	
			s_action[idx] = best_actionGPU(s_s + idx, dc_theta + iGlobal, s_Q + idx);

			// run the test using shared memory
			unsigned num_failures = 0;
			for (int tt = 0; tt < dc_test_reps; tt++) {
				take_action(s_action[idx], s_s + idx, s_s + idx, BLOCK_SIZE);
				if (terminal_state(s_s + idx, BLOCK_SIZE)) {
					++num_failures;
					randomize_state(s_s + idx, s_seeds + idx, BLOCK_SIZE);
				}
				s_action[idx] = best_actionGPU(s_s + idx, dc_theta + iGlobal, s_Q + idx);
			}
			unsigned iTest = (t-1) / dc_test_interval;
			results[iGlobal + iTest * dc_agents] = num_failures;
			
			// restore agent state
			COPY_STATE_TO_SHARED(idx, iGlobal);
		}
		if (t == dc_end_time) break;
		
		float reward = take_action(s_action[idx], s_s + idx, s_s + idx, BLOCK_SIZE);
		unsigned fail = terminal_state(s_s + idx, BLOCK_SIZE);
		if (fail) randomize_state(s_s + idx, s_seeds + idx, BLOCK_SIZE);			
		float Q_a = s_Q[idx + s_action[idx] * BLOCK_SIZE];
		s_action[idx] = choose_actionGPU(s_s + idx, dc_theta + iGlobal, s_Q + idx, s_seeds + idx);
		float Q_a_prime = s_Q[idx + s_action[idx] * BLOCK_SIZE];
		float delta = reward - Q_a + (fail ? 0 : dc_gamma * Q_a_prime);
		update_thetasGPU(dc_theta + iGlobal, dc_e + iGlobal, delta);
		if (fail) reset_traceGPU(dc_e + iGlobal);
		update_stored_QGPU(s_Q + idx, s_s + idx, dc_theta + iGlobal);
		update_traceGPU(s_action[idx], s_s + idx, dc_e + iGlobal);
	}
	COPY_STATE_TO_GLOBAL(idx, iGlobal);
}

/*
	set all eligibility trace values to 0.0f
*/
__global__ void pole_clear_trace_kernel()
{
	unsigned iGlobal = threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
	if (iGlobal < dc_agents * dc_num_features * dc_num_actions) dc_e[iGlobal] = 0.0f;
}

/*
	Do a learning session for specified number of steps.
	On entry, the theta values are valid from prior learning episodes.
	e values should all be set to 0
	
		First, randomize the state,
		Then repeat the learning process for specified number of iterations
	
	Ending state is not saved.
*/
__global__ void pole_learn_kernel(unsigned steps, unsigned first_time)
{
	unsigned iGlobal = threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
	unsigned idx = threadIdx.x;
	if (iGlobal >= dc_agents) return;
	
	__shared__ float s_s[4 * BLOCK_SIZE];
	__shared__ unsigned s_action[BLOCK_SIZE];
	__shared__ unsigned s_seeds[4 * BLOCK_SIZE];
	__shared__ float s_Q[2*BLOCK_SIZE];

	COPY_STATE_TO_SHARED(idx, iGlobal);
	
	// randomize state, determine first action and update eligibility trace
//	randomize_state(s_s + idx, s_seeds + idx, BLOCK_SIZE);
	s_action[idx] = choose_actionGPU(s_s + idx, dc_theta + iGlobal, s_Q + idx, s_seeds + idx);
	update_traceGPU(s_action[idx], s_s + idx, dc_e + iGlobal);

	// loop through specified number of time steps
	for (int t = 0; t < steps; t++) {		
		float reward = take_action(s_action[idx], s_s + idx, s_s + idx, BLOCK_SIZE);
		unsigned fail = terminal_state(s_s + idx, BLOCK_SIZE);
		if (fail) randomize_state(s_s + idx, s_seeds + idx, BLOCK_SIZE);			
		float Q_a = s_Q[idx + s_action[idx] * BLOCK_SIZE];
		s_action[idx] = choose_actionGPU(s_s + idx, dc_theta + iGlobal, s_Q + idx, s_seeds + idx);
		float Q_a_prime = s_Q[idx + s_action[idx] * BLOCK_SIZE];
		float delta = reward - Q_a + (fail ? 0 : dc_gamma * Q_a_prime);
		update_thetasGPU(dc_theta + iGlobal, dc_e + iGlobal, delta);
		if (fail) reset_traceGPU(dc_e + iGlobal);
		update_stored_QGPU(s_Q + idx, s_s + idx, dc_theta + iGlobal);
		update_traceGPU(s_action[idx], s_s + idx, dc_e + iGlobal);
	}
	COPY_STATE_TO_GLOBAL(idx, iGlobal);
}

__global__ void pole_test_kernel(float *results)
{
	unsigned iGlobal = threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
	unsigned idx = threadIdx.x;
	if (iGlobal >= dc_agents) return;
	
	__shared__ float s_s[4 * BLOCK_SIZE];
	__shared__ unsigned s_action[BLOCK_SIZE];
	__shared__ unsigned s_seeds[4 * BLOCK_SIZE];
	__shared__ float s_Q[2*BLOCK_SIZE];
	
	COPY_STATE_TO_SHARED(idx, iGlobal);
	
//	s_action[idx] = best_actionGPU(s_s + idx, dc_theta + iGlobal, s_Q + idx);

	// run the test using shared memory
	unsigned num_failures = 0;
	for (int t = 0; t < dc_test_reps; t++) {
		take_action(s_action[idx], s_s + idx, s_s + idx, BLOCK_SIZE);
		if (terminal_state(s_s + idx, BLOCK_SIZE)) {
			++num_failures;
			randomize_state(s_s + idx, s_seeds + idx, BLOCK_SIZE);
		}
		s_action[idx] = best_actionGPU(s_s + idx, dc_theta + iGlobal, s_Q + idx);
	}
	results[iGlobal] = num_failures;
	
	// restore agent state
//	COPY_STATE_TO_GLOBAL(idx, iGlobal);
}

void run_GPU(RESULTS *r)
{
#ifdef VERBOSE
	printf("\n==============================================\nRunning on GPU...\n");
#endif

	// on entry the device constant pointers have been initialized to agent's theta, 
	// eligibility trace, and state values

#ifdef DUMP_INITIAL_AGENTS
//	dump_agents_GPU("initial agents on GPU", ag);
#endif
	
	// setup constant memory on device
	set_constant_params(_p);
	
	// allocate an array to hold individual thread test results
	float *d_results = device_allocf(_p.agents * _p.num_tests);
	
	// one thread for each agent in each trial
	dim3 blockDim(BLOCK_SIZE);
	dim3 gridDim(1 + (_p.agents - 1) / BLOCK_SIZE);
	if (gridDim.x > 65535){
		gridDim.y = 1 + (gridDim.x-1) / 65535;
		gridDim.x = 1 + (gridDim.x-1) / gridDim.y;
	}
	
	dim3 clearTraceBlockDim(512);
	dim3 clearTraceGridDim(1 + (_p.agents * _p.num_features * _p.num_actions - 1) / 512);
	if (clearTraceGridDim.x > 65535) {
		clearTraceGridDim.y = 1 + (clearTraceGridDim.x-1) / 65535;
		clearTraceGridDim.x = 1 + (clearTraceGridDim.x-1) / clearTraceGridDim.y;
	}
#ifdef VERBOSE
	printf("%d total agents\n", _p.agents);
	printf("%d threads per block, (%d x %d) grid of blocks\n", blockDim.x, gridDim.x, gridDim.y);
	printf("for clearing trace: %d threads per block, (%d x %d) grid of blocks\n", 
						clearTraceBlockDim.x, clearTraceGridDim.x, clearTraceGridDim.y);
#endif
	unsigned timer;
	CREATE_TIMER(&timer);
	START_TIMER(timer);
	
	for (int i = 0; i < max(1, _p.num_tests); i++) {
#ifdef VERBOSE
		printf("[%6d] About to run pole_clear_trace_kernel ... ", i);
#endif
		pole_clear_trace_kernel<<<clearTraceGridDim, clearTraceBlockDim>>>();
	CUT_CHECK_ERROR("pole_clear_trace_kernel execution failed");
#ifdef VERBOSE
		printf("done.\n");
		printf("         About to run pole_learn_kernel for %d steps ... ", _p.test_interval);
#endif
		pole_learn_kernel<<<gridDim, blockDim>>>(_p.test_interval, i==0);
	CUT_CHECK_ERROR("pole_learn_kernel execution failed");
#ifdef VERBOSE
		printf("done.\n");
#endif

#ifdef VERBOSE
		printf("         About to run pole_test_kernel (test #%d) ... ", i);
#endif
		pole_test_kernel<<<gridDim, blockDim>>>(d_results + i * _p.agents);
		CUT_CHECK_ERROR("pole_test_kernel execution failed");
#ifdef VERBOSE
		printf("done.\n");
#endif
	}
	hipDeviceSynchronize();
	STOP_TIMER(timer, "run pole kernel on GPU");
	
//	// Check if kernel execution generated an error
//	CUT_CHECK_ERROR("Kernel execution failed");
	
	START_TIMER(timer);
	// reduce the result array on the device and copy back to the host
	row_reduce(d_results, _p.agents, _p.num_tests);
	for (int i = 0; i < _p.num_tests; i++) {
		CUDA_SAFE_CALL(hipMemcpy(r->avg_fail + i, d_results + i * _p.agents, sizeof(float), 
																hipMemcpyDeviceToHost));
		r->avg_fail[i] /= _p.trials;
	}
	hipDeviceSynchronize();
	STOP_TIMER(timer, "reduce GPU results and copy data back to host");
	
#ifdef DUMP_TERMINAL_AGENT_STATE
	dump_agents_GPU("--------------------------------------\n       Ending Agent States\n", 0);
#endif

	if (d_results) hipFree(d_results);
}

