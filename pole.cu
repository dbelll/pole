#include "hip/hip_runtime.h"
//
//  pole.cu
//  pole
//
//  Created by Dwight Bell on 8/18/10.
//  Copyright dbelll 2010. All rights reserved.
//

#include <hip/hip_runtime.h>
#include "cutil.h"

#include "cuda_rand.cu"

#include "pole.h"
#include "cuda_utils.h"
#include "main.h"

/*
	Procedures for setting up and running the pole balancing experiements on CPU and GPU
 */

static PARAMS _p;
static unsigned g_seeds[4] = {2784565659u, 1491908209u, 3415062841u, 3293636241u};

#pragma mark CPU & GPU

// random number in an interval from -max to +max using random normal with standard deviation = sd
__host__ __device__ float random_interval(unsigned *seeds, unsigned stride, float max, float sd)
{
	float r;
	// keep generating values until one is within -max to +max
	do {
		r = RandNorm(seeds, stride) / (sd * max);
	} while (r < -max || r > max);
	return r;
}

// randomize the state
__host__ __device__ void randomize_state(float *s, unsigned *seeds, unsigned stride)
{
	s[0] = random_interval(seeds, stride, ANGLE_MAX, ANGLE_MAX / SD_FOR_MAX);
	s[stride] = random_interval(seeds, stride, ANGLE_VEL_MAX, ANGLE_VEL_MAX / SD_FOR_MAX);
	s[2*stride] = random_interval(seeds, stride, X_MAX, X_MAX / SD_FOR_MAX);
	s[3*stride] = random_interval(seeds, stride, X_VEL_MAX, X_VEL_MAX / SD_FOR_MAX);
}

// reset eligibility traces to 0.0f
__host__ __device__ void reset_trace(float *e, unsigned num_features, unsigned num_actions, 
										unsigned stride)
{
	for (int f = 0; f < num_features; f++) {
		for (int a = 0; a < num_actions; a++) {
			e[(a + f * num_actions) * stride] = 0.0f;
		}
	}
}

__device__ __host__ unsigned terminal_state(float *s, unsigned stride)
{
	return s[2*stride] < X_MIN || s[2*stride] > X_MAX || 
			s[0] < ANGLE_MIN || s[0] > ANGLE_MAX;
}

// take an action from the current state, s, returning the reward and saving the new state in s_prime
__device__ __host__ float take_action(unsigned a, float *s, float *s_prime, unsigned stride)
{
	// formulas are from: Brownlee. The pole balancing problem: a benchmark control theory problem. hdl.handle.net (2005)
	
	// determine force from the action
	float F = a ? FORCE : -FORCE;

	float ang = s[0];
	float ang_vel = s[stride];
	float cos_a = cos(ang);
	float sin_a = sin(ang);
	
	// calculate angular acceleration
	float ang_accel = GRAV * sin_a;
	ang_accel += cos_a * (-F - POLE_MASS * POLE_LENGTH * ang_vel * ang_vel * sin_a) / 
							(CART_MASS + POLE_MASS);
	ang_accel /= POLE_LENGTH * (4.0f/3.0f - POLE_MASS * cos_a * cos_a / (CART_MASS + POLE_MASS));
	
	float x = s[2*stride];
	float x_vel = s[3*stride];

	// calculate x acceleration
	float x_accel = F + POLE_MASS * POLE_LENGTH * (ang_vel * ang_vel * sin_a - ang_accel * cos_a);
	x_accel /= (CART_MASS + POLE_MASS);
	
	// update ang, ang_vel and x, x_vel
	s_prime[0] = ang + TAU * ang_vel;
	s_prime[stride] = ang_vel + TAU * ang_accel;
	s_prime[2*stride] = x + TAU * x_vel;
	s_prime[3*stride] = x_vel + TAU * x_accel;
	
	// determine the reward
	float reward = terminal_state(s_prime, stride) ? REWARD_FAIL : REWARD_NON_FAIL;
	
	return reward;
}

//float take_action_debug(unsigned a, float *s, float *s_prime, unsigned stride)
//{
//	// formulas are from: Brownlee. The pole balancing problem: a benchmark control theory problem. hdl.handle.net (2005)
//	// determine force from the action
//	float F = a ? FORCE : -FORCE;
//
//	float ang = s[0];
//	float ang_vel = s[stride];
//	float x = s[2*stride];
//	float x_vel = s[3*stride];
//
//	float cos_a = cos_a(ang);
//	float sin_a = sin_a(ang);
//	
//	printf("[tack_action_debug] action=%d, angle=%7.4f, angle_vel=%7.4f\n", a, 
//
//	// calculate angular acceleration
//	float ang_accel = GRAV * sin_a;
//	ang_accel += cos_a * (-F - POLE_MASS * POLE_LENGTH * ang_vel * ang_vel * sin_a) / 
//							(CART_MASS + POLE_MASS);
//	ang_accel /= POLE_LENGTH * (4.0f/3.0f - POLE_MASS * cos_a * cos_a / (CART_MASS + POLE_MASS));
//	
//	// calculate x acceleration
//	float x_accel = F + POLE_MASS * POLE_LENGTH * (ang_vel * ang_vel * sin_a - ang_accel * cos_a);
//	x_accel /= (CART_MASS + POLE_MASS);
//	
//	// update ang, ang_vel and x, x_vel
//	s_prime[0] = ang + TAU * ang_vel;
//	s_prime[1] = ang_vel + TAU * ang_accel;
//	s_prime[2] = x + TAU * x_vel;
//	s_prime[3] = x_vel + TAU * x_accel;
//	
//	// determine the reward
//	float reward = REWARD_NON_FAIL;
//	if (s_prime[2] < X_MIN || s_prime[2] > X_MAX || 
//		s_prime[0] < ANGLE_MIN || s_prime[0] > ANGLE_MAX) 
//	{
//		reward = REWARD_FAIL;
//	}
//	
//	return reward;
//}

// Calculate which feature division the state value falls into, based on the min, max,
// and number of divisions.
__device__ __host__ unsigned feature_val_for_state_val(float s, float minv, float maxv, 
														unsigned div)
{
	return max(0, min(div-1, (unsigned)((s-minv)/(maxv-minv) * (float)div)));
}

// Determine which feature corresponds to the given state
__device__ __host__ unsigned feature_for_state(float *s, unsigned stride)
{
	unsigned feature = feature_val_for_state_val(s[0], ANGLE_MIN, ANGLE_MAX, ANGLE_DIV);
	feature += (ANGLE_DIV) * 
				feature_val_for_state_val(s[stride], ANGLE_VEL_MIN, ANGLE_VEL_MAX, ANGLE_VEL_DIV);
	feature += (ANGLE_DIV * ANGLE_VEL_DIV) * 
				feature_val_for_state_val(s[2 * stride], X_MIN, X_MAX, X_DIV);
	feature += (ANGLE_DIV * ANGLE_VEL_DIV * X_DIV) * 
				feature_val_for_state_val(s[3 * stride], X_VEL_MIN, X_VEL_MAX, X_VEL_DIV);
	return feature;
}

__device__ __host__ const char * failure_type(float *s, unsigned stride)
{
	if (s[0] < ANGLE_MIN) return "Angle < MIN";
	if (s[0] > ANGLE_MAX) return "Angle > MAX";
	if (s[2] < X_MIN) return "X < MIN";
	if (s[2] > X_MAX) return "X > MAX";
	return "";
}

// Calculate a number with the division for each state variable
__device__ __host__ unsigned divs_for_feature(unsigned feature)
{
	unsigned divs = feature % ANGLE_DIV;
	feature /= ANGLE_DIV;
	divs += 16 * (feature % ANGLE_VEL_DIV);
	feature /= ANGLE_VEL_DIV;
	divs += 256 * (feature % X_DIV);
	feature /= X_DIV;
	divs += 4096 * feature;
	return divs;
}

// calculate the Q value for an action from a state
__device__ __host__ float calc_Q(float *s, unsigned a, float *theta, unsigned stride, 
																			unsigned num_actions)
{
	// only one feature corresponds with any given state
	unsigned feature = feature_for_state(s, stride);
	float Q = theta[(a + feature * num_actions) * stride];
	return Q;
}

__device__ __host__ void update_stored_Q(float *Q, float *s, float *theta, unsigned stride, 
																			unsigned num_actions)
{
	for (int a = 0; a < num_actions; a++) {
		Q[a * stride] = calc_Q(s, a, theta, stride, num_actions);
	}
}

// Calculate the Q value for each action from the given state, storing the values in Q
// Return the action with the highest Q value
__device__ __host__ unsigned best_action(float *s, float *theta, float *Q, unsigned stride,
										 unsigned num_actions)
{
	// calculate the Q value for each action
	Q[0] = calc_Q(s, 0, theta, stride, num_actions);
	unsigned best_action = 0;
	float bestQ = Q[0];

	for (int a = 1; a < num_actions; a++) {
		Q[a * stride] = calc_Q(s, a, theta, stride, num_actions);
		if (Q[a * stride] > bestQ) {
			bestQ = Q[a * stride];
			best_action = a;
		}
	}
	return best_action;
}

// choose action from current state, storing Q values for each possible action in Q
__device__ __host__ unsigned choose_action(float *s, float *theta, float epsilon, unsigned stride, 
											float *Q, unsigned num_actions, unsigned *seeds)
{
	// always calcualte the best action and store all the Q values for each action
	unsigned a = best_action(s, theta, Q, stride, num_actions);
	if (RandUniform(seeds, stride) < epsilon){
		// choose random action
		float r = RandUniform(seeds, stride);
		a = r * num_actions;
	}
	return a;
}

// Update eligibility traces based on action and state
__device__ __host__ void update_trace(unsigned action, float *s, float *e, unsigned num_features,
										unsigned num_actions, unsigned stride, float gamma, float lambda)
{
	unsigned feature = feature_for_state(s, stride);
	for (int f = 0; f < num_features; f++) {
		for (int a = 0; a < num_actions; a++) {
			unsigned index = (a + f * num_actions) * stride;
			// Replacing trace with optional block
			if (f == feature) {
				// set to 1.0 for action selected from current state,
				// set to 0.0 for actions not taken from current state
				e[index] = (a == action) ? 1.0f : 0.0f;
			}else {
				// decay all other values
				e[index] *= gamma * lambda;
			}
		}
	}
}

// Update theta values for one agent
// theta = theta + alpha * delta * eligibility trace
//__device__
 __host__ void update_thetas(float *theta, float *e, float alpha, float delta, unsigned num_features, unsigned stride, unsigned num_actions)
{
	if (alpha == 0.0f || delta == 0.0f) return;
#ifdef DUMP_THETA_UPDATE_CALCULATIONS
	printf("updating thetas for alpha = %9.6f, delta = %9.6f\n", alpha, delta);
#endif
	for (int fa = 0; fa < num_features * num_actions; fa++) {
		if (e[fa*stride] > 0.001f) {
#ifdef DUMP_THETA_UPDATE_CALCULATIONS
			printf("   feature-action %5d(%4x) %3d with trace %9.6f changed from %9.6f", (fa/num_actions), divs_for_feature(fa/num_actions), (fa%num_actions), e[fa*stride], theta[fa*stride]);
#endif
			theta[fa * stride] += alpha * delta * e[fa * stride];
#ifdef DUMP_THETA_UPDATE_CALCULATIONS
			printf(" to %9.6f\n", theta[fa*stride]);
#endif
		}
	}
}

#pragma mark -
#pragma mark CPU

void set_params(PARAMS p){ _p = p;}

void dump_agent(AGENT_DATA *ag, unsigned agent)
{
	printf("[agent %d]: ", agent);
	printf("   seeds = %u, %u, %u, %u\n", ag->seeds[agent], ag->seeds[agent + _p.agents], 
									   ag->seeds[agent + 2*_p.agents], ag->seeds[agent + 3*_p.agents]);
#ifdef AGENT_DUMP_INCLUDE_THETA_E
	printf("FEATURE       ACTION    THETA       E  \n");
	for (int f = 0; f < _p.num_features; f++) {
		for (int action = 0; action < _p.num_actions; action++) {
			printf("%7d %4x %7d %9.6f %9.6f\n", f, divs_for_feature(f), action, 
				   ag->theta[agent + (action + f * _p.num_actions) * _p.agents], 
				   ag->e[agent + (action + f * _p.num_actions) * _p.agents]);
		}
	}
#endif
	printf("   angle    angleV       x         xV        Q0        Q1   feature\n");
	unsigned feature = feature_for_state(ag->s + agent, _p.agents);
	printf("%9.6f %9.6f %9.6f %9.6f %9.6f %9.6f %7d(%4x)\n", ag->s[agent], ag->s[agent + _p.agents], ag->s[agent + 2*_p.agents], ag->s[agent + 3*_p.agents], ag->Q[agent], ag->Q[agent + _p.agents],
		feature, divs_for_feature(feature));
	
	printf("ACTION  Q-value\n");
//		printf("number of actions is %d\n", p.num_actions);
	for (int action = 0; action < _p.num_actions; action++) {
		(action == ag->action[agent]) ? printf("-->") : printf("   ");
		printf("%3d  %9.6f\n", action, ag->Q[agent + action * _p.agents]);
	}
	printf("\n");
}

void dump_agents(const char *str, AGENT_DATA *ag)
{
	printf("%s, agents for %s\n", str, ag->device_flag ? "device" : "host");
	for (int agent = 0; agent < _p.agents; agent++) {
		dump_agent(ag, agent);
	}
}

// generate random seeds for the sepecified number of agents
unsigned *create_seeds(unsigned num_agents)
{
	unsigned *seeds = (unsigned *)malloc(num_agents * 4 * sizeof(unsigned));
	for (int i = 0; i < num_agents * 4; i++) {
		seeds[i] = RandUniformui(g_seeds, 1);
	}
	return seeds;
}

// create wgts set initially to random values between RAND_WGT_MIN and RAND_WGT_MAX
float *create_theta(unsigned num_agents, unsigned num_features, unsigned num_actions)
{
#ifdef VERBOSE
	printf("create_theta for %d agents and %d features\n", num_agents, num_features);
#endif
	float *theta = (float *)malloc(num_agents * num_features * num_actions * sizeof(float));
	for (int i = 0; i < num_agents * num_features * num_actions; i++) {
//		float r = RandUniform(g_seeds, 1);
//		theta[i] = (RAND_WGT_MAX - RAND_WGT_MIN) * r + RAND_WGT_MIN;
//		printf("randome = %7.4f, theta = %7.4f\n", r, theta[i]);
		theta[i] = (RAND_WGT_MAX - RAND_WGT_MIN) * RandUniform(g_seeds, 1) + RAND_WGT_MIN;
	}
	return theta;
}

// initial eligibility traces to 0.0f
float *create_e(unsigned num_agents, unsigned num_features, unsigned num_actions)
{
#ifdef VERBOSE
	printf("create_e for %d agents and %d features and %d actions\n", num_agents, num_features, num_actions);
#endif
	float *e = (float *)malloc(num_agents * num_features * num_actions * sizeof(float));
	for (int i = 0; i < num_agents * num_features * num_actions; i++) {
		e[i] = 0.0f;
	}
	return e;
}

// initial random states
float *create_states(unsigned num_agents, unsigned *seeds)
{
	float *states = (float *)malloc(num_agents * NUM_STATE_VALUES * sizeof(float));
	for (int i = 0; i < num_agents; i++) {
//		states[i] = random_interval(seeds + i, num_agents, ANGLE_MAX, STATE_SD);
//		states[i + num_agents] = random_interval(seeds+i, num_agents, ANGLE_VEL_MAX, STATE_SD);
//		states[i + 2 * num_agents] = random_interval(seeds+i, num_agents, X_MAX, STATE_SD);
//		states[i + 3 * num_agents] = random_interval(seeds+i, num_agents, X_VEL_MAX, STATE_SD);
		randomize_state(states + i, seeds + i, num_agents);
	}
	return states;
}

RESULTS *initialize_results()
{
#ifdef VERBOSE
	printf("initializing result arrays...\n");
#endif
	RESULTS *r = (RESULTS *)malloc(sizeof(RESULTS));
	r->begun = (float *)malloc(_p.data_lines * sizeof(float));
	r->ended = (float *)malloc(_p.data_lines * sizeof(float));
	r->total_length = (float *)malloc(_p.data_lines * sizeof(float));
	return r;
}

void free_results(RESULTS *r)
{
#ifdef VERBOSE
	printf("freeing result arrays...\n");
#endif
	if (r) {
		if (r->begun) free(r->begun);
		if (r->ended) free(r->ended);
		if (r->total_length) free(r->total_length);
		free(r);
	}
}

void display_results(const char *str, RESULTS *r)
{
	printf("%s \n", str);
	printf("    TIME    BEGUN   ENDED   TOT_LENGTH\n");
	for (int i = 0; i < _p.data_lines; i++) {
		printf("   [%4d] %7.0f %7.0f %12.0f\n", i, r->begun[i], r->ended[i], r->total_length[i]);
	}
}

unsigned *create_actions(unsigned num_agents, unsigned num_actions)
{
	unsigned *actions = (unsigned *)malloc(num_agents * num_actions * sizeof(unsigned));
	for (int i = 0; i < num_agents * num_actions; i++) {
		actions[i] = num_actions;	// not possible action
	}
	return actions;
}

// Initialize agents on the CPU.  Some values will be re-used for GPU agents
AGENT_DATA *initialize_agentsCPU()
{
#ifdef VERBOSE
	printf("initializing agents on CPU...\n");
#endif
	AGENT_DATA *ag = (AGENT_DATA *)malloc(sizeof(AGENT_DATA));
	ag->seeds = create_seeds(_p.agents);
	ag->theta = create_theta(_p.agents, _p.num_features, _p.num_actions);
	ag->e = create_e(_p.agents, _p.num_features, _p.num_actions);
	unsigned rows = _p.agents * ((_p.state_size + 2) * _p.sharing_interval + _p.state_size + 1);
	ag->ep_data = (float *)malloc(rows * sizeof(float));
	ag->s = create_states(_p.agents, ag->seeds);
	ag->Q = (float *)malloc(_p.agents * _p.num_actions * sizeof(float));
	ag->action = create_actions(_p.agents, _p.num_actions);
//	ag->prev_action = (unsigned *)malloc(_p.agents * sizeof(unsigned));
//	ag->f_prev_state = (unsigned *)malloc(_p.agents * sizeof(unsigned));
	return ag;
}

void dump_state(float *s, unsigned stride)
{
	printf("(%9.6f,%9.6f,%9.6f,%9.6f)[%d]\n", s[0], s[stride], s[2*stride], s[3*stride], 
															feature_for_state(s, stride));
}

// run tests for all agents and return the average failures
float run_test(AGENT_DATA *ag)
{
	unsigned num_failures = 0;
	
	// initialize all agent states
	for (int agent = 0; agent < _p.agents; agent++) {
//		printf("agent %d before testing...\n", agent);
//		dump_agent(ag, agent);
		
		// save agent state
		float s0 = ag->s[agent];
		float s1 = ag->s[agent + _p.agents];
		float s2 = ag->s[agent + 2*_p.agents];
		float s3 = ag->s[agent + 3*_p.agents];
		unsigned act = ag->action[agent];
		unsigned seed0 = ag->seeds[agent];
		unsigned seed1 = ag->seeds[agent + _p.agents];
		unsigned seed2 = ag->seeds[agent + 2 * _p.agents];
		unsigned seed3 = ag->seeds[agent + 3 * _p.agents];
		float Q0 = ag->Q[agent];
		float Q1 = ag->Q[agent + _p.agents];
		
		randomize_state(ag->s + agent, ag->seeds + agent, _p.agents);

		// run the test for specified number of reps
		for (int t = 0; t < _p.test_reps; t++) {
			take_action(ag->action[agent], ag->s+agent, ag->s+agent, _p.agents);
			if (terminal_state(ag->s + agent, _p.agents)){
				++num_failures;
				randomize_state(ag->s + agent, ag->seeds + agent, _p.agents);
			}
			// choose action with epsilon = 0.0
			ag->action[agent] = choose_action(ag->s + agent, ag->theta + agent, 0.0f, 
									_p.agents, ag->Q + agent, _p.num_actions, ag->seeds + agent);
		}
		
		// restore agent state
		ag->s[agent] = s0;
		ag->s[agent + _p.agents] = s1;
		ag->s[agent + 2*_p.agents] = s2;
		ag->s[agent + 3*_p.agents] = s3;
		act = ag->action[agent] = act;
		ag->seeds[agent] = seed0;
		ag->seeds[agent + _p.agents] = seed1;
		ag->seeds[agent + 2 * _p.agents] = seed2;
		ag->seeds[agent + 3 * _p.agents] = seed3;
		ag->Q[agent] = Q0;
		ag->Q[agent + _p.agents] = Q1;
		
//		printf("after testing...\n");
//		dump_agent(ag, agent);
	}



	return num_failures / (float)_p.agents;
}

void run_CPU_noshare(AGENT_DATA *ag, RESULTS *r)
{
	unsigned tot_fails = 0;
#ifdef DUMP_INTERMEDIATE_FAIL_COUNTS
	unsigned prev_tot_fails = 0;
#endif
#ifdef VERBOSE
	printf(" no sharing\n");
#endif

	// on entry the agent's theta, eligibility trace, and state values have been initialized
	
#ifdef DUMP_AGENT_ACTIONS
		printf("-----------------------------------------------------------\n");
		printf("---------------------- INITIAL SETUP ----------------------\n");
		printf("-----------------------------------------------------------\n");
#endif
//	float orig_a = ag->s[0];
//	float orig_aV = ag->s[_p.agents];
//	float orig_x = ag->s[2*_p.agents];
//	float orig_xV = ag->s[3*_p.agents];
//	printf("orig state: %6.2f %6.2f %6.2f %6.2f\n", orig_a, orig_aV, orig_x, orig_xV);

	// set-up agents to begin the loop by choosing the first action and updating traces
	for (int agent = 0; agent < _p.agents; agent++) {
		ag->action[agent] = choose_action(ag->s + agent, ag->theta + agent, _p.epsilon, _p.agents,
										ag->Q + agent, _p.num_actions, ag->seeds + agent);

#ifdef DUMP_AGENT_ACTIONS
		printf("agent %d will choose action %d from state ", agent, ag->action[agent]);
		dump_state(ag->s + agent, _p.agents);
#endif

		update_trace(ag->action[agent], ag->s + agent, ag->e + agent, _p.num_features, 
												_p.num_actions, _p.agents, _p.gamma, _p.lambda);		
	}

#ifdef DUMP_AGENT_ACTIONS
	printf("----------------------------------------------------\n");
	printf("-------------- BEGIN MAIN LOOP ---------------------\n");
	printf("----------------------------------------------------\n");
#endif	

	// main loop, repeat for the number of trials
	for (int t = 0; t < _p.time_steps; t++) {
	
		if (0 == (t % _p.test_interval) && (t > 0)) {
			printf("*********[%3d] test results =%7.2f\n", t / _p.test_interval, run_test(ag));
		}

#ifdef DUMP_AGENT_ACTIONS
	printf("\n------------------ TIME STEP%3d ------------------------\n", t);
#endif	

		for (int agent = 0; agent < _p.agents; agent++) {

			// stored state is s      stored Q's are Q(s)  
			
#ifdef DUMP_AGENT_ACTIONS
			printf("<<<<<<<< AGENT %d >>>>>>>>>>>>\n", agent);
			printf("time step %d, agent %d ready for next action\n", t, agent);
			dump_agent(ag, agent);
#endif
			// take the action already chosen and saved in ag->action
			unsigned prev_feature = feature_for_state(ag->s, _p.agents);
			float reward = take_action(ag->action[agent], ag->s + agent, ag->s + agent, _p.agents);

#ifdef DUMP_AGENT_BRIEF
			(agent == 0) ? printf("[step%4d]", t) : printf("          ");
			printf("[agent%3d] took action:%2d, got reward:%6.3f, new state is ", agent, ag->action[agent], reward);
			dump_state(ag->s + agent, _p.agents);
#endif
			
			// stored state is s_prime      stored Q's are Q(s)
			unsigned fail = terminal_state(ag->s + agent, _p.agents);
			if (fail){
#ifdef DUMP_FAILURE_TIMES
				printf("Agent%4d Failure at %d taking action %d from state %d (%x) resulting in %s\n", agent, t, ag->action[agent], prev_feature, divs_for_feature(prev_feature), failure_type(ag->s + agent, _p.agents));
#endif
#ifdef DUMP_AGENT_STATE_ON_FAILURE
				printf("session initial state was angle=%6.2f,  angleV=%6.2f, x=%6.2f, xV=%6.2f\n",
						orig_a, orig_aV, orig_x, orig_xV);
				dump_agent(ag, agent);
#endif
				randomize_state(ag->s + agent, ag->seeds + agent, _p.agents);
//				if (agent == 0){
//					orig_a = ag->s[0];
//					orig_aV = ag->s[_p.agents];
//					orig_x = ag->s[2*_p.agents];
//					orig_xV = ag->s[3*_p.agents];
//				}
				++tot_fails;
			}
						

			float Q_a = ag->Q[agent + ag->action[agent] * _p.agents];

#ifdef DUMP_AGENT_ACTIONS
			if (fail) printf("-------------------------------------------------------\n!!!! terminal state reached, next state is random\n---------------------------------------------------\n\n");
			printf("agent %d, took action %d, got reward %6.3f, now in state s_prime = " , agent,	ag->action[agent], reward);
			dump_state(ag->s + agent, _p.agents);
#endif

#ifdef DUMP_CALCULATIONS
			printf("reward is %9.6f, Q[%d] for state s is %9.6f\n", reward, ag->action[agent], Q_a);
#endif

//			ag->prev_action[agent] = ag->action[agent];
//			ag->f_prev_state[agent] = feature_for_state(ag->s + agent, _p.agents);
			ag->action[agent] = choose_action(ag->s + agent, ag->theta + agent, _p.epsilon,
								_p.agents, ag->Q + agent, _p.num_actions, ag->seeds + agent);
			
			// Stored Q values are now based on the new state, s_prime

#ifdef DUMP_AGENT_ACTIONS
			printf("agent %d's next action will be %d with Q-value %9.6f\n", agent, ag->action[agent], ag->Q[agent + ag->action[agent] * _p.agents]);
//			dump_state(ag->s + agent, _p.agents);
#endif

			float Q_a_prime = ag->Q[agent + ag->action[agent] * _p.agents];
			float delta = reward - Q_a + (fail ? 0 : _p.gamma * Q_a_prime);

#ifdef DUMP_CALCULATIONS
			printf("discount is %9.6f, newQ[%d] is %9.6f, so delta is %9.6f\n", _p.gamma, 
												ag->action[agent], (fail ? 0.0f : Q_a_prime), delta);
#endif

#ifdef DUMP_AGENT_ACTIONS
			printf("[update_theta]:\n");
#endif

			update_thetas(ag->theta + agent, ag->e + agent, _p.alpha, delta, _p.num_features,
																	 _p.agents, _p.num_actions);
			if (fail) reset_trace(ag->e + agent, _p.num_features, _p.num_actions, _p.agents);

			update_stored_Q(ag->Q + agent, ag->s + agent, ag->theta + agent, _p.agents, 
																				_p.num_actions);
			
#ifdef DUMP_AGENT_ACTIONS
			printf("[update_trace]\n");
#endif

			update_trace(ag->action[agent], ag->s + agent, ag->e + agent, _p.num_features, _p.num_actions, _p.agents, _p.gamma, _p.lambda);
			
#ifdef DUMP_AGENT_ACTIONS
//			printf("agent state after updating theta and eligibility trace:\n");
//			dump_agent(ag, agent);
#endif
		}

#ifdef DUMP_INTERMEDIATE_FAIL_COUNTS
		if (0 == (1+t) % DUMP_INTERMEDIATE_FAIL_COUNTS) {
			printf("intermediate fail count =%7.2f\n", (tot_fails - prev_tot_fails)/(float)_p.trials);
			prev_tot_fails = tot_fails;
		}
#endif


	}
	
	printf("*********[%3d] test results =%7.2f\n", _p.time_steps / _p.test_interval, run_test(ag));

#ifdef DUMP_TERMINAL_AGENT_STATE
	printf("\n==================================================\n");
	printf(  "               ENDING AGENT STATES");
	for (int agent = 0; agent < _p.agents; agent++) {
		dump_agent(ag, agent);

#ifdef AGENT_TERMINAL_DUMP_INCLUDE_THETA_E
		printf("FEATURE       ACTION    THETA       E  \n");
		for (int f = 0; f < _p.num_features; f++) {
			for (int action = 0; action < _p.num_actions; action++) {
				printf("%7d %4x %7d %9.6f %9.6f\n", f, divs_for_feature(f), action, 
					   ag->theta[agent + (action + f * _p.num_actions) * _p.agents], 
					   ag->e[agent + (action + f * _p.num_actions) * _p.agents]);
			}
		}
#endif

	}
#endif		
	printf("total failures = %d\n", tot_fails);
}

void run_CPU_share(AGENT_DATA *cv, RESULTS *r)
{
#ifdef VERBOSE
	printf(" sharing in agent blocks of %d\n", _p.agent_group_size);
#endif

}

void run_CPU(AGENT_DATA *cv, RESULTS *r)
{
#ifdef VERBOSE
	printf("running on CPU...");
#endif
	
	_p.block_sharing ? run_CPU_share(cv, r) : run_CPU_noshare(cv, r);	
}

void free_agentsCPU(AGENT_DATA *ag)
{
#ifdef VERBOSE
	printf("freeing agents on CPU...\n");
#endif
	if (ag) {
		if (ag->seeds) free(ag->seeds);
		if (ag->theta) free(ag->theta);
		if (ag->e) free(ag->e);
		if (ag->ep_data) free(ag->ep_data);
		if (ag->s) free(ag->s);
		if (ag->Q) free(ag->Q);
		free(ag);
	}
}

#pragma mark -
#pragma mark GPU

AGENT_DATA *initialize_agentsGPU(AGENT_DATA *agCPU)
{
#ifdef VERBOSE
	printf("initializing agents on GPU...\n");
#endif
	AGENT_DATA *ag = (AGENT_DATA *)malloc(sizeof(AGENT_DATA));
	
	return ag;	
}

void free_agentsGPU(AGENT_DATA *ag)
{
#ifdef VERBOSE
	printf("freeing agents on GPU...\n");
#endif
	if (ag) {
		if (ag->seeds) hipFree(ag->seeds);
		if (ag->theta) hipFree(ag->theta);
		if (ag->e) hipFree(ag->e);
		if (ag->ep_data) hipFree(ag->ep_data);
		if (ag->s) hipFree(ag->s);
		if (ag->Q) hipFree(ag->Q);
		free(ag);
	}
}

void run_GPU(AGENT_DATA *cv, RESULTS *r)
{
#ifdef VERBOSE
	printf("running on CPU...\n");
#endif
}

__global__ void kernel_operation(int n, float *x)
{
	
}

void gpu_operation(int n, float *x)
{
	unsigned int timer;
	CREATE_TIMER(&timer);

	// copy data to device
	START_TIMER(timer);
	float *d_x = NULL;
	int size = n * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, size));
	CUDA_SAFE_CALL(hipMemcpy(d_x, x, size, hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	STOP_TIMER(timer, "copy data to device");


	// calculate size of block and grid
	int tot_threads = 32 * (1 + (n-1)/32);	// smallest multiple of 32 >= n
	dim3 blockDim(min(512,tot_threads));
	dim3 gridDim(1 + (tot_threads -1)/512, 1);
	if (gridDim.x > 65535) {
		gridDim.y = 1 + (gridDim.x-1) / 65535;
		gridDim.x = 1 + (gridDim.x-1) / gridDim.y;
	}
	
	// Do the operation on the device
	START_TIMER(timer);	
	kernel_operation<<<gridDim, blockDim>>>(n, d_x);
	hipDeviceSynchronize();
	STOP_TIMER(timer, "GPU operations");

	// Check if kernel execution generated an error
	CUT_CHECK_ERROR("Kernel execution failed");
	
	
	// copy results back to host
	START_TIMER(timer);
	CUDA_SAFE_CALL(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
	STOP_TIMER(timer, "copy data back to host");
	
	
}

